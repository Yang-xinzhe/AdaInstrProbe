
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

__global__ void print_from_gpu(void) {
    printf("Hello World! from thread [%d, %d] from device\n", threadIdx.x, blockIdx.x);
}

int main(int argc, char const *argv[])
{
    printf("Hello world from Host\n");
    while(1){
        print_from_gpu<<<1,1>>>();
        sleep(1);
    }
    

    hipDeviceSynchronize();
    return 0;
}

